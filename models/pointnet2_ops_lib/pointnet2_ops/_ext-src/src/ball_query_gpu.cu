#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// input: new_xyz(b, m, 4) xyz(b, n, 4)
// output: idx(b, m, nsample)
__global__ void query_ball_point_kernel(int b, int n, int m, float radius,
                                        int nsample,
                                        const float *__restrict__ new_xyz,
                                        const float *__restrict__ xyz,
                                        int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * 4;
  new_xyz += batch_index * m * 4;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    float new_x = new_xyz[j * 4 + 0];
    float new_y = new_xyz[j * 4 + 1];
    float new_z = new_xyz[j * 4 + 2];
    float new_q = new_xyz[j * 4 + 3];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      float x = xyz[k * 4 + 0];
      float y = xyz[k * 4 + 1];
      float z = xyz[k * 4 + 2];
      float q = xyz[k * 4 + 3];
      float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z) + (new_q - q) * (new_q - q);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

void query_ball_point_kernel_wrapper(int b, int n, int m, float radius,
                                     int nsample, const float *new_xyz,
                                     const float *xyz, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  query_ball_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, nsample, new_xyz, xyz, idx);

  CUDA_CHECK_ERRORS();
}
